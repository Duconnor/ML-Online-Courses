#include "hip/hip_runtime.h"
/* CUDA blur
 * Kevin Yuh, 2014 */

#include <cstdio>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_convolve.cuh"


/* 
Atomic-max function. You may find it useful for normalization.

We haven't really talked about this yet, but __device__ functions not
only are run on the GPU, but are called from within a kernel.

Source: 
http://stackoverflow.com/questions/17399119/
cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
*/
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}



__global__
void
cudaProdScaleKernel(const hipfftComplex *raw_data, const hipfftComplex *impulse_v, 
    hipfftComplex *out_data,
    int padded_length) {


    /* TODO: Implement the point-wise multiplication and scaling for the
    FFT'd input and impulse response. 

    Recall that these are complex numbers, so you'll need to use the
    appropriate rule for multiplying them. 

    Also remember to scale by the padded length of the signal
    (see the notes for Question 1).

    As in Assignment 1 and Week 1, remember to make your implementation
    resilient to varying numbers of threads.

    */
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    while (idx < padded_length) {
        out_data[idx].x = (raw_data[idx].x * impulse_v[idx].x - raw_data[idx].y * impulse_v[idx].y) / padded_length;
        out_data[idx].y = (raw_data[idx].x * impulse_v[idx].y + raw_data[idx].y * impulse_v[idx].x) / padded_length;
        idx += blockDim.x * gridDim.x;
    }
}

__global__
void
cudaMaximumKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the maximum-finding.

    There are many ways to do this reduction, and some methods
    have much better performance than others. 

    For this section: Please explain your approach to the reduction,
    including why you chose the optimizations you did
    (especially as they relate to GPU hardware).

    You'll likely find the above atomicMax function helpful.
    (CUDA's atomicMax function doesn't work for floating-point values.)
    It's based on two principles:
        1) From Week 2, any atomic function can be implemented using
        atomic compare-and-swap.
        2) One can "represent" floating-point values as integers in
        a way that preserves comparison, if the sign of the two
        values is the same. (see http://stackoverflow.com/questions/
        29596797/can-the-return-value-of-float-as-int-be-used-to-
        compare-float-in-cuda)

    */

    extern __shared__ float shared_memory_data[];
    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int shared_idx = threadIdx.x;
    int start_idx = blockIdx.x * blockDim.x;
    while (start_idx < padded_length) {
        // Initialize the shared memory
        shared_memory_data[shared_idx] = 0.0;
        __syncthreads();

        // Copy data from global memory to the shared memory
        if (global_idx < padded_length) {
            shared_memory_data[shared_idx] = out_data[global_idx].x > 0 ? out_data[global_idx].x : -out_data[global_idx].x;
        }
        __syncthreads();

        // Begin reduction
        int shared_idx_threshold = blockDim.x;
        for (int i = 0; i < (int)log2((double)blockDim.x); i++) {
            shared_idx_threshold /= 2;
            if (shared_idx < shared_idx_threshold) {
                shared_memory_data[shared_idx] = 
                    shared_memory_data[shared_idx] > shared_memory_data[shared_idx + shared_idx_threshold] ? shared_memory_data[shared_idx] : shared_memory_data[shared_idx + shared_idx_threshold];
            }
            __syncthreads();
        }

        if (shared_idx == 0) {
            // Use the maximum value inside this block to update the global maximum abs val
            atomicMax(max_abs_val, shared_memory_data[0]);
        }

        // Update the start_idx
        start_idx += blockDim.x * gridDim.x;
        global_idx += blockDim.x * gridDim.x;
    }

}

__global__
void
cudaDivideKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the division kernel. Divide all
    data by the value pointed to by max_abs_val. 

    This kernel should be quite short.
    */

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    while (idx > padded_length) {
        out_data[idx].x = out_data[idx].x / (*max_abs_val);
        idx += blockDim.x * gridDim.x;
    }
}


void cudaCallProdScaleKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const hipfftComplex *raw_data,
        const hipfftComplex *impulse_v,
        hipfftComplex *out_data,
        const unsigned int padded_length) {
        
    /* TODO: Call the element-wise product and scaling kernel. */
    cudaProdScaleKernel<<<dim3(blocks), dim3(threadsPerBlock)>>>(raw_data, impulse_v, out_data, padded_length);
}

void cudaCallMaximumKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        

    /* TODO 2: Call the max-finding kernel. */
    cudaMaximumKernel<<<dim3(blocks), dim3(threadsPerBlock), threadsPerBlock * sizeof(float)>>>(out_data, max_abs_val, padded_length);
}


void cudaCallDivideKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        
    /* TODO 2: Call the division kernel. */
    cudaDivideKernel<<<dim3(blocks), dim3(threadsPerBlock)>>>(out_data, max_abs_val, padded_length);
}
