/**
 * CUDA Point Alignment
 * George Stathopoulos, Jenny Lee, Mary Giambrone, 2019*/ 

#include <cstdio>
#include <stdio.h>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "hip/hip_runtime_api.h"
#include <string>
#include <fstream>

#include "obj_structures.h"

// hip/hip_runtime_api.h contains the error checking macros. note that they're called
// CUDA_CALL, CUBLAS_CALL, and CUSOLVER_CALL instead of the previous names

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char *argv[]) {

    if (argc != 4)
    {
        printf("Usage: ./point_alignment [file1.obj] [file2.obj] [output.obj]\n");
        return 1;
    }

    std::string filename, filename2, output_filename;
    filename = argv[1];
    filename2 = argv[2];
    output_filename = argv[3];

    std::cout << "Aligning " << filename << " with " << filename2 <<  std::endl;
    Object obj1 = read_obj_file(filename);
    std::cout << "Reading " << filename << ", which has " << obj1.vertices.size() << " vertices" << std::endl;
    Object obj2 = read_obj_file(filename2);

    std::cout << "Reading " << filename2 << ", which has " << obj2.vertices.size() << " vertices" << std::endl;
    if (obj1.vertices.size() != obj2.vertices.size())
    {
        printf("Error: number of vertices in the obj files do not match.\n");
        return 1;
    }

    ///////////////////////////////////////////////////////////////////////////
    // Loading in obj into vertex Array
    ///////////////////////////////////////////////////////////////////////////

    int point_dim = 4; // 3 spatial + 1 homogeneous
    int num_points = obj1.vertices.size();

    // in col-major
    float * x1mat = vertex_array_from_obj(obj1);
    float * x2mat = vertex_array_from_obj(obj2);

    // for (int i = 0; i < num_points; i++) {
    //     for (int j = 0; j < point_dim; j++) {
    //         std::cout << x1mat[i + j * num_points] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    ///////////////////////////////////////////////////////////////////////////
    // Point Alignment
    ///////////////////////////////////////////////////////////////////////////

    // TODO: Initialize cublas handle
    hipblasHandle_t handle;

    hipblasCreate(&handle);
    // DONE

    float * dev_x1mat;
    float * dev_x2mat;
    float * dev_xx4x4;
    float * dev_x1Tx2;

    // TODO: Allocate device memory and copy over the data onto the device
    // Hint: Use hipblasSetMatrix() for copying

    hipMalloc((void**)&dev_x1mat, sizeof(float) * point_dim * num_points);
    hipMalloc((void**)&dev_x2mat, sizeof(float) * point_dim * num_points);
    hipMalloc((void**)&dev_xx4x4, sizeof(float) * point_dim * point_dim);
    hipMalloc((void**)&dev_x1Tx2, sizeof(float) * point_dim * point_dim);

    hipblasSetMatrix(num_points, point_dim, sizeof(float), x1mat, num_points, dev_x1mat, num_points);
    hipblasSetMatrix(num_points, point_dim, sizeof(float), x2mat, num_points, dev_x2mat, num_points);
    // DONE

    // Now, proceed with the computations necessary to solve for the linear
    // transformation.

    float one = 1;
    float zero = 0;

    // TODO: First calculate xx4x4 and x1Tx2
    // Following two calls should correspond to:
    //   xx4x4 = Transpose[x1mat] . x1mat
    //   x1Tx2 = Transpose[x1mat] . x2mat

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, point_dim, point_dim, num_points, &one, dev_x1mat, num_points, dev_x1mat, num_points, &zero, dev_xx4x4, point_dim);
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, point_dim, point_dim, num_points, &one, dev_x1mat, num_points, dev_x2mat, num_points, &zero, dev_x1Tx2, point_dim);
    // DONE

    // TODO: Finally, solve the system using LU-factorization! We're solving
    //         xx4x4 . m4x4mat.T = x1Tx2   i.e.   m4x4mat.T = Inverse[xx4x4] . x1Tx2
    //
    //       Factorize xx4x4 into an L and U matrix, ie.  xx4x4 = LU
    //
    //       Then, solve the following two systems at once using cusolver's getrs
    //           L . temp  =  P . x1Tx2
    //       And then then,
    //           U . m4x4mat = temp
    //
    //       Generally, pre-factoring a matrix is a very good strategy when
    //       it is needed for repeated solves.

    // TODO: Make handle for cuSolver
    hipsolverHandle_t solver_handle;

    hipsolverDnCreate(&solver_handle);
    // DONE


    // TODO: Initialize work buffer using hipsolverDnSgetrf_bufferSize
    float * work;
    int Lwork;

    hipsolverDnSgetrf_bufferSize(solver_handle, point_dim, point_dim, dev_xx4x4, point_dim, &Lwork);
    // DONE

    // TODO: compute buffer size and prepare memory

    hipMalloc((void**)&work, sizeof(float) * Lwork);
    // DONE

    // TODO: Initialize memory for pivot array, with a size of point_dim
    int * pivots;

    hipMalloc((void**)&pivots, sizeof(int) * point_dim);
    // DONE

    int *info;


    // TODO: Now, call the factorizer hipsolverDnSgetrf, using the above initialized data

    hipMalloc((void**)&info, sizeof(int));
    hipsolverDnSgetrf(solver_handle, point_dim, point_dim, dev_xx4x4, point_dim, work, pivots, info);
    // DONE

    // TODO: Finally, solve the factorized version using a direct call to hipsolverDnSgetrs

    hipsolverDnSgetrs(solver_handle, HIPBLAS_OP_N, point_dim, point_dim, dev_xx4x4, point_dim, pivots, dev_x1Tx2, point_dim, info);
    // DONE

    // TODO: Destroy the cuSolver handle

    hipsolverDnDestroy(solver_handle);
    // DONE

    // TODO: Copy final transformation back to host. Note that at this point
    // the transformation matrix is transposed
    float * out_transformation;

    out_transformation = (float*)malloc(sizeof(float) * point_dim * point_dim);
    hipMemcpy(out_transformation, dev_x1Tx2, sizeof(float) * point_dim * point_dim, hipMemcpyDeviceToHost);    
    // DONE

    // TODO: Don't forget to set the bottom row of the final transformation
    //       to [0,0,0,1] (right-most columns of the transposed matrix)

    // Although the obtained transformation matrix is transposed, it is stored in
    // column major manner, therefore when we copy it back to host, the matrix
    // is transposed again and becomes normal

    // NOTE: Why we need to set the bottom row to [0,0,0,1] here?
    // It is true in theory that the computed transformation will have
    // [0,0,0,1] in the bottom row because the last coordianate of all
    // points is 1 and unchanged. However, due to numerical issues, it might becomes
    // [3.4e-8, 0, 0, 1] stuffs like this, so it's better to set it mannually.
    for (int i = 0; i < point_dim - 1; i++) {
        out_transformation[(point_dim - 1) * point_dim + i] = 0.0;
    }
    out_transformation[(point_dim - 1) * point_dim + point_dim - 1] = 1.0;
    // DONE

    // Print transformation in row order.
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            std::cout << out_transformation[i * point_dim + j] << " ";
        }
        std::cout << "\n";
    }

    ///////////////////////////////////////////////////////////////////////////
    // Transform point and print output object file
    ///////////////////////////////////////////////////////////////////////////

    // TODO Allocate and Initialize data matrix
    float * dev_pt;

    hipMalloc((void**)&dev_pt, sizeof(float) * point_dim * num_points);
    hipblasSetMatrix(num_points, point_dim, sizeof(float), x1mat, num_points, dev_pt, num_points);
    // DONE

    // TODO Allocate and Initialize transformation matrix
    float * dev_trans_mat;

    hipMalloc((void**)&dev_trans_mat, sizeof(float) * point_dim * point_dim);
    hipblasSetMatrix(point_dim, point_dim, sizeof(float), out_transformation, point_dim, dev_trans_mat, point_dim);
    // DONE

    // TODO Allocate and Initialize transformed points
    float * dev_trans_pt;

    hipMalloc((void**)&dev_trans_pt, sizeof(float) * num_points * point_dim);
    // DONE

    float one_d = 1;
    float zero_d = 0;

    // TODO Transform point matrix
    //          (4x4 trans_mat) . (nx4 pointzx matrix)^T = (4xn transformed points)

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, point_dim, num_points, point_dim, &one_d, dev_trans_mat, point_dim, dev_pt, num_points, &zero_d, dev_trans_pt, point_dim);
    // DONE

    // So now dev_trans_pt has shape (4 x n)
    float * trans_pt;

    trans_pt = (float*)malloc(sizeof(float) * point_dim * num_points);
    hipMemcpy(trans_pt, dev_trans_pt, sizeof(float) * point_dim * num_points, hipMemcpyDeviceToHost);

    // get Object from transformed vertex matrix
    Object trans_obj = obj_from_vertex_array(trans_pt, num_points, point_dim, obj1);

    // print Object to output file
    std::ofstream obj_file (output_filename);
    print_obj_data(trans_obj, obj_file);

    // free CPU memory
    free(trans_pt);

    ///////////////////////////////////////////////////////////////////////////
    // Free Memory
    ///////////////////////////////////////////////////////////////////////////

    // TODO: Free GPU memory

    hipFree(dev_x1mat);
    hipFree(dev_x2mat);
    hipFree(dev_xx4x4);
    hipFree(dev_x1Tx2);
    hipFree(work);
    hipFree(pivots);
    hipFree(dev_pt);
    hipFree(dev_trans_mat);
    hipFree(dev_trans_pt);
    hipblasDestroy(handle);

    // TODO: Free CPU memory
    free(out_transformation);
    free(x1mat);
    free(x2mat);

}

